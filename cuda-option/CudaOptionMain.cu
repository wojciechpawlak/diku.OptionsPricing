
#include "Version1.cuh"
#include "Version2.cuh"
#include "Version3.cuh"
#include "../common/Args.hpp"

using namespace std;
using namespace trinom;

cuda::CudaRuntime run(const Valuations &valuations, vector<real> &results,
    const int version, const int blockSize, const SortType sortType, const bool isTest)
{
    switch (version)
    {
    case 1:
    {
        cuda::option::KernelRunNaive kernelRun;
        kernelRun.run(valuations, results, blockSize, sortType, isTest);
        return kernelRun.runtime;
    }
    case 2:
    {
        cuda::option::KernelRunCoalesced kernelRun;
        kernelRun.run(valuations, results, blockSize, sortType, isTest);
        return kernelRun.runtime;
    }
    case 3:
    {
        cuda::option::KernelRunCoalescedGranular kernelRun(blockSize);  // block-level padding granularity
        kernelRun.run(valuations, results, blockSize, sortType, isTest);
        return kernelRun.runtime;
    }
    case 4:
    {
        cuda::option::KernelRunCoalescedGranular kernelRun(32);         // warp-level padding granularity
        kernelRun.run(valuations, results, blockSize, sortType, isTest);
        return kernelRun.runtime;
    }
    }
    return cuda::CudaRuntime();
}

void computeValuations(const Valuations &valuations, const int version,
    const int blockSize, const SortType sortType, const int runs, const bool isTest)
{
    if (isTest)
    {
        cout << "Cuda one valuation per thread version " << version << endl;
    }

    if (runs > 0)
    {
        if (isTest)
        {
            cout << "Performing " << runs << " runs..." << endl;
        }
        cuda::CudaRuntime best;
        for (auto i = 0; i < runs; ++i)
        {
            vector<real> results;
            results.resize(valuations.ValuationCount);
            auto runtime = run(valuations, results, version, blockSize, sortType, isTest);
            if (runtime < best)
            {
                best = runtime;
            }
        }
        if (isTest)
        {
            cout << "Best times: kernel " << best.KernelRuntime << " microsec, total " << best.TotalRuntime << " microsec." << endl;
        }
        else
        {
            cout << version << ',' << blockSize << ',' << (char)sortType << ',' << best.KernelRuntime << ',' << best.TotalRuntime << ',' << best.DeviceMemory << endl;
        }
    }
    else
    {
        vector<real> results;
        results.resize(valuations.ValuationCount);
        run(valuations, results, version, blockSize, sortType, isTest);

        if (!isTest)
        {
            Arrays::write_array(cout, results);
        }
    }
}

int main(int argc, char *argv[])
{
    Args args(argc, argv);

    if (args.test)
    {
        cout << "Loading valuations " << args.valuations << endl;
    }

    // Read valuations and yield curve.
    Valuations valuations(args.valuations);

    // Initialize cuda device.
    hipSetDevice(args.device);
    hipFree(0);

    for (auto &version : args.versions)
    {
        for (auto &blockSize : args.blockSizes)
        {
            for (auto &sortType : args.sorts)
            {
                computeValuations(valuations, version, blockSize, sortType, args.runs, args.test);
            }
        }
    }

    return 0;
}
